#define __KERNEL__
//#include "shared_test.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/ioctl.h>
#include <dlfcn.h>
#include <errno.h>
#include <uvm_ioctl.h>
//#include "mmap_cu.h"
#include <sys/time.h>

#include <dirent.h>
#include "ucm_mmap.h"

#define UCM_ERR(fmt, ...) \
	printf("UCM_ERR: %s(): " fmt, __func__, ##__VA_ARGS__);

#define UCM_DBG(fmt, ...) \
	printf("UCM_DBG: %s(): " fmt, __func__, ##__VA_ARGS__);


#define MAP_FAILED ((void*)-1)
#define MAP_HUGETLB	0x100000	/* create a huge page mapping */
typedef void *(*orig_mmap_f_type)(void *addr, size_t length, int prot,
			int flags, int fd, off_t offset);
typedef int (*orig_munmap_f_type)(void *addr, size_t length);

// This will output the proper CUDA error strings in the event that a 
// CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if(hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, 
        				line, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

struct file_map_struct
{
	void *cuda_ptr;
	void *cpu_ptr;
	int taken;
};

#define SUPPORTED_FILES_MAPPED 10
static struct file_map_struct mappings_arr[SUPPORTED_FILES_MAPPED];
static int mappings_arr_idx = 0;
static int mappings_cnt = 0;

#define NVIDI_UVM_CHAR_DEV "/dev/nvidia-uvm"
int nvidia_uvm_fd = -1;
static int open_nvidia_old(void)
{	
	nvidia_uvm_fd = open(NVIDI_UVM_CHAR_DEV, O_RDWR);
	if (nvidia_uvm_fd < 0){
		UCM_ERR("Failed opening %s err = %d\n", NVIDI_UVM_CHAR_DEV, nvidia_uvm_fd);
		return -1;
	}
	return 0;
}

#define PSF_DIR "/proc/self/fd"
static int open_nvidia(void)
{
	DIR *d;
	d = opendir(PSF_DIR);
	char psf_path[256];
	char *psf_realpath;
	struct dirent *dir;

	if (d)
	{
		while ((dir = readdir(d)) != NULL)
		{
			if (dir->d_type == DT_LNK)
			{
				sprintf(psf_path, "%s/%s", PSF_DIR, dir->d_name);
				psf_realpath = realpath(psf_path, NULL);
				if (strcmp(psf_realpath, NVIDI_UVM_CHAR_DEV) == 0)
					nvidia_uvm_fd = atoi(dir->d_name);
				free(psf_realpath);
				if (nvidia_uvm_fd >= 0)
					break;
			}
		}
		closedir(d);
	}
	if (nvidia_uvm_fd < 0)
	{
		fprintf(stderr, "Cannot open %s\n", PSF_DIR);
		return -1;
	}
	return 0;
}

static void close_nvidia(void)
{	
	close(nvidia_uvm_fd);
	nvidia_uvm_fd = -1;
}

static int map_vma_ioctl(unsigned long long uvm_base,
						 unsigned long long cpu_base, int map)
{
	UVM_MAP_VMA_RANGE_PARAMS params;
	UVM_UNMAP_VMA_RANGE_PARAMS uparams;
	
	params.uvm_base = uvm_base;
	params.cpu_base = cpu_base;
	
	uparams.uvm_base = uvm_base;
	
	if (map) {
		if (ioctl(nvidia_uvm_fd, UVM_MAP_VMA_RANGE, &params) == -1) {
    			UCM_ERR("ioctl to uvm failed\n");
    			return -1;
    		}
	} else {
		if (ioctl(nvidia_uvm_fd, UVM_UNMAP_VMA_RANGE, &uparams) == -1) {
            		UCM_ERR("ioctl to uvm failed\n");
            		return -1;
        	}
	}
	return 0;
}
static int touch_pages(unsigned long long uvm_base, unsigned long length)
{
	UVM_TOUCH_RANGE_PARAMS params;
	char c;

	params.uvm_base = uvm_base;
	params.start_addr = uvm_base;
	params.length = length;
	if (ioctl(nvidia_uvm_fd, UVM_TOUCH_RANGE, &params) == -1) {
                UCM_ERR("ioctl to uvm failed with code %u\n", params.rmStatus);
        	return -1;
	}
	return 0;
}

void *map_file_on_gpu(void *addr, size_t length, int prot, int flags,
                  int fd, off_t offset)
{
	long int pagenum=0;
	bool prefetch_to_cpu = false;
	orig_mmap_f_type orig_mmap;
	orig_mmap = (orig_mmap_f_type)dlsym(RTLD_NEXT,"mmap");

	struct file_map_struct *mapping = &mappings_arr[mappings_arr_idx++];
	
	if (flags & PREFETCH_TO_CPU) {
		prefetch_to_cpu = true;
		flags &= NOT_PREFETCH_TO_CPU;
		printf("got PREFETCH_TO_CPU. revert it: flags = 0x%lx\n", flags);
	}
	if (mapping->taken) {
		UCM_ERR("mapping taken!!\n");
		//TODO: handle this case/ a lock might be needed
	}
	mapping->taken = 1;
	mappings_cnt++;
	mapping->cpu_ptr = 0;

	mapping->cpu_ptr = orig_mmap(addr, length, prot, flags , fd, offset);
	if (mapping->cpu_ptr == MAP_FAILED)
	{
	 	printf("Oh dear, something went wrong with orig_mmap()! %s, errno=%d, length=%lu\n", strerror(errno), errno, length);
	    	exit(EXIT_FAILURE);
	}

	long start1, end1;
	struct timeval timecheck1;
	gettimeofday(&timecheck1, NULL);
	start1 = (long)timecheck1.tv_sec * 1000 + (long)timecheck1.tv_usec / 1000;
	checkCudaErrors( hipMallocManaged((void **)&mapping->cuda_ptr, length, hipMemAttachGlobal) );
	gettimeofday(&timecheck1, NULL);
	end1 = (long)timecheck1.tv_sec * 1000 + (long)timecheck1.tv_usec / 1000;
    printf("Time_consumed_malloc,%ld,ms\n", (end1 - start1));

	/* Need to try open the file after first hipMalloc. Otherwise it's not created */
	if (nvidia_uvm_fd < 0 && open_nvidia()) {
        	//TODO: Handle this err properly
        	return NULL;
    	}

	/* Now issue IOCTL to uvm to set up the connection in uvm_va_space */
	if (map_vma_ioctl((unsigned long long)mapping->cuda_ptr,
					  (unsigned long long)mapping->cpu_ptr, 1)) {
    		UCM_ERR("ioctl to uvm failed\n");
    		//TODO: handle properly
    		return NULL;
    	}
	
	if (!prefetch_to_cpu && touch_pages((unsigned long long)mapping->cuda_ptr, length)) {
                UCM_ERR("touch pages failed\n");
       }
#if 0
if (!touch_data) {
	checkCudaErrors( hipMemPrefetchAsync(mapping->cuda_ptr, length, -1, 0) );
	UCM_DBG("hipMemPrefetchAsync to host\n");
	checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors( hipGetLastError() );
} else {
#endif
	if (1 /*length / 4096 >= 48*/) {
		long start, end;
    		struct timeval timecheck;
 		gettimeofday(&timecheck, NULL);
    		start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
		while (pagenum * 4096 < length) {
			char *tmp = (char *)(mapping->cuda_ptr) + pagenum * 4096;
			if (*tmp == '!')
				UCM_ERR("hit :) %d\n", pagenum);
			pagenum++;
		}
		gettimeofday(&timecheck, NULL);
    		end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
 		printf("touch_pages,%ld,ms\n", (end - start));
	}

	if (!prefetch_to_cpu && touch_pages((unsigned long long)mapping->cuda_ptr, length)) {
                UCM_ERR("touch pages second time failed\n");
        }
	return mapping->cuda_ptr;
}

extern "C"
void *mmap(void *addr, size_t length, int prot, int flags,
                  int fd, off_t offset)
{
	orig_mmap_f_type orig_mmap;
	orig_mmap = (orig_mmap_f_type)dlsym(RTLD_NEXT,"mmap");
	
	if (!(flags & MAP_ON_GPU))
		return orig_mmap(addr, length, prot, flags, fd, offset);

	//If I got here MAP_ON_GPU is on
	if (flags & ACQUIRE) {
		//This is a hack for calling aquire
		flags |= MA_PROC_NVIDIA;
		return (void *)maquire(addr, length, flags);
	}
	if (flags & RELEASE) {
		//This is a hack for calling release
		flags |= MA_PROC_NVIDIA;
		return (void *)mrelease(addr, length, flags);
	}
    return map_file_on_gpu(addr, length, prot, flags, fd, offset);
}

void *gmmap(void *addr, size_t length, int prot, int flags,
                  int fd, off_t offset) {
	return mmap(addr, length, prot, flags, fd, offset);
}

extern "C"
int munmap (void *addr, size_t length)
{
	int i;
	orig_munmap_f_type orig_munmap;
	orig_munmap = (orig_munmap_f_type)dlsym(RTLD_NEXT,"munmap");

	for (i = 0; i < mappings_cnt; i++)
		if (mappings_arr[i].cuda_ptr == addr && mappings_arr[i].taken) {
			struct file_map_struct *mapping = &mappings_arr[i];
			mappings_arr[i].taken = 0;
			mappings_cnt--;

			/* Now issue IOCTL to uvm to remove the connection in uvm_va_space */
			if (map_vma_ioctl((unsigned long long)mapping->cuda_ptr,
						  (unsigned long long)mapping->cpu_ptr, 0)) {
				UCM_ERR("ioctl to uvm failed\n");
				//TODO: handle properly
			}
			//giving an err. need to understand why	
			checkCudaErrors( hipFree(addr) );

			if (!mappings_cnt)
                close_nvidia();
			UCM_DBG("call original munmap for cpu addr\%p. free managed memory\n", mapping->cpu_ptr);
			return orig_munmap(mapping->cpu_ptr, length);
		}
    return orig_munmap(addr, length);
}

int gmunmap (void *addr, size_t length) {
	return munmap(addr, length);
}

//map the pages into cpu_ptr vma
int maquire(void *start, size_t length, int flags) {
	int i;
	long int pagenum=0;
	struct file_map_struct *mapping = NULL;

	for (i = 0; i < mappings_cnt; i++)
		if (mappings_arr[i].cuda_ptr == start && mappings_arr[i].taken) {
			mapping = &mappings_arr[i];
		}
	if (!mapping)
		return -1;

	if (0 /*length / 4096 >= 48*/) {
		long start, end;
			struct timeval timecheck;
		gettimeofday(&timecheck, NULL);
			start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
		while (pagenum * 4096 < length) {
			char *tmp2 = (char *)(mapping->cpu_ptr) + pagenum * 4096;
			if (*tmp2 == '!')
				UCM_ERR("hit :) %d\n", pagenum);
			pagenum++;
		}
		gettimeofday(&timecheck, NULL);
			end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
	}

	return syscall(327, start, length, 0x10);
}

int mrelease(void *start, size_t len, int flags) {
	return syscall(328, start, len, flags);
}

